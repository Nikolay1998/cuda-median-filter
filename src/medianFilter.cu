#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "MedianFilter.h"
#include <time.h>
#define TILE_SIZE 4 

__global__ void medianFilterKernel(unsigned char *inputImageKernel, unsigned char *outputImagekernel, int imageWidth, int imageHeight)
{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char filterVector[9] = { 0,0,0,0,0,0,0,0,0 };   //Take fiter window
	if ((row == 0) || (col == 0) || (row == imageHeight - 1) || (col == imageWidth - 1))
		outputImagekernel[row*imageWidth + col] = 0; //Deal with boundry conditions
	else {
		for (int x = 0; x < WINDOW_SIZE; x++) {
			for (int y = 0; y < WINDOW_SIZE; y++) {
				filterVector[x*WINDOW_SIZE + y] = inputImageKernel[(row + x - 1)*imageWidth + (col + y - 1)];   // setup the filterign window.
			}
		}
		for (int i = 0; i < 9; i++) {
			for (int j = i + 1; j < 9; j++) {
				if (filterVector[i] > filterVector[j]) {
					//Swap the variables.
					char tmp = filterVector[i];
					filterVector[i] = filterVector[j];
					filterVector[j] = tmp;
				}
			}
		}
		outputImagekernel[row*imageWidth + col] = filterVector[4];   //Set the output variables.
	}
}

bool MedianFilterGPU(Bitmap* image, Bitmap* outputImage) {
	//Cuda error and image values.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipError_t status;
	int width = image->Width();
	int height = image->Height();

	int size = width * height * sizeof(char);
	//initialize images.
	unsigned char *deviceinputimage;
	hipMalloc((void**)&deviceinputimage, size);
	status = hipGetLastError();
	if (status != hipSuccess) {
		std::cout << "Kernel failed for hipMalloc : " << hipGetErrorString(status) <<
			std::endl;
		return false;
	}
	hipMemcpy(deviceinputimage, image->image, size, hipMemcpyHostToDevice);
	status = hipGetLastError();
	if (status != hipSuccess) {
		std::cout << "Kernel failed for hipMemcpy hipMemcpyHostToDevice: " << hipGetErrorString(status) <<
			std::endl;
		hipFree(deviceinputimage);
		return false;
	}
	unsigned char *deviceOutputImage;
	hipMalloc((void**)&deviceOutputImage, size);
	//take block and grids.
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)image->Width() / (float)TILE_SIZE),
		(int)ceil((float)image->Height() / (float)TILE_SIZE));

	medianFilterKernel <<< dimGrid, dimBlock >>> (deviceinputimage, deviceOutputImage, width, height);

	// save output image to host.
	hipMemcpy(outputImage->image, deviceOutputImage, size, hipMemcpyDeviceToHost);
	status = hipGetLastError();

	if (status != hipSuccess) {
		std::cout << "Kernel failed for hipMemcpy hipMemcpyDeviceToHost: " << hipGetErrorString(status) <<
			std::endl;
		hipFree(deviceinputimage);
		hipFree(deviceOutputImage);
		return false;
	}
	//Free the memory
	hipFree(deviceinputimage);
	hipFree(deviceOutputImage);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	printf("gputime %fms\n", time);
	return true;
}